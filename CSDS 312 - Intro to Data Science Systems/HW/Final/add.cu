#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void vector_add_single(float *a, float *b, float *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    float *a, *b, *c;
    float *d_a, *d_b, *d_c;

    size_t bytes = N * sizeof(float);
    hipHostMalloc(&a, bytes, hipHostMallocDefault);
    hipHostMalloc(&b, bytes, hipHostMallocDefault);
    hipHostMalloc(&c, bytes, hipHostMallocDefault);

    for (int i = 0; i < N; i++) {
        a[i] = i * 1.0f;
        b[i] = (N - i) * 1.0f;
    }

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);

    vector_add_single<<<1, 1>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    hipHostFree(a); hipHostFree(b); hipHostFree(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
